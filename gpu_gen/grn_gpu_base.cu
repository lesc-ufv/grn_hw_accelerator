#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define NUM_STATES (1<<20)
#define NUM_COPYS  (1 << 10)
#define NUM_NOS 70

typedef unsigned int uint32_t;

using namespace std; 

__device__
bool state_comp(bool *s0, bool *s1, int size){
   for(int i = 0; i < size; i++){
       if(s0[i] != s1[i])return false;
   }
   return true;
}

__device__
void pass(bool *State){
  
  bool currentState [NUM_NOS];
  for(int i = 0; i < NUM_NOS; i++){
       currentState[i] = State[i];
  }
  State[0] = currentState[41];
  State[1] = currentState[21];
  State[2] = currentState[41] && ! currentState[33] ;
  State[3] = currentState[36] && ! ( currentState[37] || currentState[7] ) ;
  State[4] = ( ( currentState[51] || currentState[34] ) && currentState[37] ) && ! currentState[3] ;
  State[5] = ! ( currentState[21] && currentState[1] ) ;
  State[6] = ( currentState[49] || currentState[32] ) && ! ( currentState[34] || currentState[37] ) ;
  State[7] = ( currentState[8] || currentState[9] ) && ! currentState[22] ;
  State[8] = currentState[18] && ! ( currentState[11] || currentState[33] ) ;
  State[9] = currentState[14] && ! ( currentState[22] || currentState[33] ) ;
  State[10] = currentState[47] && ! currentState[48] ;
  State[11] = currentState[32] ;
  State[12] = currentState[43] && currentState[53] ;
  State[13] = ( currentState[5] || currentState[49] || currentState[27] ) && ! currentState[21] ;
  State[14] = currentState[31] ;
  State[15] = currentState[35] ;
  State[16] = currentState[29] ;
  State[17] = currentState[65] ;
  State[18] = currentState[53] || currentState[17] ;
  State[19] = currentState[16] ;
  State[20] = currentState[60] ;
  State[21] = ! ( currentState[15] || currentState[3] ) ;
  State[22] = ( currentState[32] || currentState[49] ) && ! currentState[44] ;
  State[23] = ! currentState[24] ;
  State[24] = ( currentState[3] || ( currentState[43] && currentState[53] ) ) ;
  State[25] = currentState[20] && ! currentState[50] ;
  State[26] = currentState[2] || currentState[30] ;
  State[27] = ( ( currentState[5] || currentState[16] ) && currentState[26] ) && ! currentState[21] ;
  State[28] = ( currentState[34] && currentState[3] ) && ! ( currentState[21] || currentState[0] ) ;
  State[29] = currentState[39] || currentState[41] ;
  State[30] = currentState[10] || currentState[52] || currentState[53] ;
  State[31] = ( currentState[4] || currentState[51] || currentState[10] ) && ! currentState[6] ;
  State[32] = ! currentState[23] ;
  State[33] = ( currentState[34] || currentState[45] ) && ! ( currentState[21] || currentState[7] ) ;
  State[34] = ( currentState[38] || currentState[26] || currentState[0] ) && ! currentState[28] ;
  State[35] = currentState[12] ;
  State[36] = ( currentState[15] || currentState[40] ) && ! currentState[38] ;
  State[37] = currentState[10] && ! currentState[3] ;
  State[38] = currentState[34] && ! ( currentState[32] || currentState[27] ) ;
  State[39] = currentState[10] || currentState[40] ;
  State[40] = currentState[15] || currentState[20] ;
  State[41] = currentState[53] && ! currentState[42] ;
  State[42] = currentState[32] || currentState[49] ;
  State[43] = currentState[48] ;
  State[44] = currentState[31] ;
  State[45] = currentState[52] && ! currentState[27] ;
  State[46] = currentState[45] || currentState[32] ;
  State[47] = currentState[34] || currentState[18] ;
  State[48] = currentState[16] || currentState[53] ;
  State[49] = currentState[25] ;
  State[50] = currentState[49] ;
  State[51] = currentState[8] && ! currentState[6] ;
  State[52] = currentState[58] && ! currentState[46] ;
  State[53] = currentState[55] ;
  State[54] = ( currentState[63] || currentState[66] ) && ! currentState[60] ;
  State[55] = currentState[59] ;
  State[56] = currentState[61] && ! ( currentState[64] || currentState[58] ) ;
  State[57] = ( currentState[62] || currentState[64] ) && ! ( currentState[63] || currentState[58] || currentState[61] ) ;
  State[58] = currentState[54] ;
  State[59] = ( currentState[64] || currentState[67] ) && ! currentState[63] ;
  State[60] = currentState[59] || currentState[66] || currentState[32] ;
  State[61] = currentState[66] || currentState[56] ;
  State[62] = currentState[66] || currentState[59] ;
  State[63] = currentState[54] || currentState[56] ;
  State[64] = currentState[57] || currentState[65] ;
  State[65] = currentState[64] && ! currentState[58] ;
  State[66] = ( currentState[67] || currentState[55] ) && ! currentState[63] ;
  State[67] = currentState[32] ;
  State[68] = ( currentState[19] && currentState[13] ) && ! ( currentState[33] || currentState[7] ) ;
  State[69] = currentState[7] ;

}

__global__
void findAttractor(bool *attractors, uint32_t * transients, uint32_t *periods, uint32_t numThreads){
   const int numNos = NUM_NOS;
   const int numState = NUM_STATES;
   const int numCopys = NUM_COPYS;
   unsigned int periodo, transient;
   bool S0[numNos];
   bool S1[numNos];
   uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   uint32_t step =  numState / numCopys; 
   uint32_t rest =  numState % numCopys;
   uint32_t begin = 0;
   uint32_t end = step - 1;
   bool flag = true;
   if(thread < numThreads){
       if(rest > 0){
           end = end + 1;
           rest = rest - 1;
       }else{
           flag = false;
       }
       
       for(uint32_t i = 0; i < numCopys;i++){
         if(i == thread) break;
         if(rest > 0){
           end = end + 1;
           begin = begin + 1;
           rest = rest - 1;
         }
         else if(rest == 0 && flag){
           begin = begin + 1;
           flag = 0;
         }
         begin += step;
         end += step;
       }
       //printf("Thread %d: begin:%d end: %d numStates: %d\n",thread,begin,end,end-begin+1);
       for(uint32_t  i = begin; i <= end; i++){
          int aux = i;
          for(int k = 0; k < numNos; k++){
              S0[k] = aux & 1 != 0;
              S1[k] = aux & 1 != 0;
	      aux >>= 1;
          }
          periodo = 0;
          transient = 0;
          do{
             pass(S0);
             pass(S1);
             pass(S1);
             transient++;
          }while (state_comp(S0,S1,numNos));
          do{
             pass(S0);
             periodo++;
          }while(state_comp(S0,S1,numNos));
          periodo--;
	  
          transients[i] = transient;
          periods[i]= periodo;
          for(int s = 0; s < numNos; s++){
            attractors[i*numNos + s] = S0[s];
	  }
       }
   }
}


int getIndice(int num_col, int i, int j)
{
    return (i*num_col)+ j;
}
/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    size_t numNos = NUM_NOS;
    size_t numState = NUM_STATES;
    size_t size = numNos * numState * sizeof(bool);
    size_t size_transients = numState*sizeof(uint32_t);
    size_t size_periods = numState*sizeof(uint32_t);
    size_t totalBytes = size+size_transients+size_periods;
    size_t kb = totalBytes/(1024);
    size_t mb = kb/(1024);
    size_t gb = mb/(1024);
    printf("Find attractors net %lu nodes in %lu initials states.\n", numNos,numState);
    printf("Memory usage: %lu Gb or %lu Mb or %lu Kb.\n",gb,mb,kb);
    uint32_t *h_transients = (uint32_t*)malloc(size_transients);
    // Verify that allocations succeeded
    if (h_transients == NULL)
    {
        fprintf(stderr, "Failed to allocate h_transients!\n");
        exit(EXIT_FAILURE);
    }    
    uint32_t *h_periods = (uint32_t*)malloc(size_periods);
    // Verify that allocations succeeded
    if (h_periods == NULL)
    {
        fprintf(stderr, "Failed to allocate h_periods!\n");
        exit(EXIT_FAILURE);
    }
    // Allocate the host output vector C
    bool *h_attractors = (bool *)malloc(size);

    // Verify that allocations succeeded
    if (h_attractors == NULL)
    {
        fprintf(stderr, "Failed to allocate h_attractors!\n");
        exit(EXIT_FAILURE);
    }
    // Allocate the device vectors
    uint32_t *d_transients = NULL;
    err = hipMalloc((void **)&d_transients, NUM_STATES*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    uint32_t *d_periods = NULL;
    err = hipMalloc((void **)&d_periods, NUM_STATES*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    bool *d_attractors = NULL;
    err = hipMalloc((void **)&d_attractors, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid =(NUM_COPYS + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    findAttractor<<<blocksPerGrid, threadsPerBlock>>>(d_attractors,d_transients,d_periods,NUM_COPYS);
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch findAttractor kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_transients, d_transients,size_transients, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_transients from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
        err = hipMemcpy(h_periods, d_periods, size_periods, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_periods from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_attractors, d_attractors, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_attractors from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*
    printf("Attractor found:\n");
    for(int i = 0; i < numState; i++){
       for(int j = 0; j < numNos; j++){
          printf("%d",h_attractors[getIndice(numNos,i,j)]);
      }
      printf("\n");
    }
    printf("\n");
    */
    err = hipFree(d_transients);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_periods);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_attractors);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_transients);
    free(h_periods);
    free(h_attractors);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}





