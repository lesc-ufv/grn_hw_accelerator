#include "hip/hip_runtime.h"
#include <grn_gpu.h>
#include <iostream>
#include <string>
#include <utility>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <iomanip>
#include <timer.h>

using namespace std;

size_t read_input_file(std::string input_file,bool **initial_states);
void write_output_file(std::string output_file,uint32_t * transients, uint32_t * periods, bool * attractors, size_t num_states);
void write_report_file(double exec_time);

TIMER_INIT(1);

int main(int argc, char *argv[]) {

    if(argc < 3){
        printf("Usage: ./%s <GRN initial states file> <Output file name>\n",argv[0]);
        return EXIT_FAILURE;
    }

    std::string input_file = argv[1];
    std::string output_file = argv[2];
    hipError_t err = hipSuccess;

    bool *h_initial_states = NULL;
    size_t num_states = read_input_file(input_file,&h_initial_states);

    size_t size = NUM_NOS * num_states * sizeof(bool);
    size_t size_transients = num_states*sizeof(uint32_t);
    size_t size_periods = num_states*sizeof(uint32_t);

    uint32_t *h_transients = (uint32_t*)malloc(size_transients);
    uint32_t *h_periods = (uint32_t*)malloc(size_periods);
    bool *h_attractors = (bool *)malloc(size);

    bool     *d_initial_states = NULL;
    uint32_t *d_transients = NULL;
    uint32_t *d_periods = NULL;
    bool     *d_attractors = NULL;

    printf("Find attractors net %d nodes in %lu initials states.\n", NUM_NOS,num_states);

    if (h_transients == NULL)
    {
        fprintf(stderr, "Failed to allocate h_transients!\n");
        exit(EXIT_FAILURE);
    }

    if (h_periods == NULL)
    {
        fprintf(stderr, "Failed to allocate h_periods!\n");
        exit(EXIT_FAILURE);
    }

    if (h_attractors == NULL)
    {
        fprintf(stderr, "Failed to allocate h_attractors!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_initial_states, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_transients, num_states*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_periods, num_states*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_attractors, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    TIMER_START(0);

    err = hipMemcpy(d_initial_states,h_initial_states,size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_initial_states from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid =(num_states + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    findAttractor<<<blocksPerGrid, threadsPerBlock>>>(d_initial_states, d_attractors,d_transients,d_periods,num_states);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch findAttractor kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_transients, d_transients,size_transients, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_transients from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_periods, d_periods, size_periods, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_periods from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_attractors, d_attractors, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_attractors from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    TIMER_STOP_ID(0);

    err = hipFree(d_initial_states);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_initial_states (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_transients);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_periods);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_attractors);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    write_output_file(output_file,h_transients,h_periods,h_attractors,num_states);
    write_report_file(TIMER_REPORT_MS(0));

    free(h_initial_states);
    free(h_transients);
    free(h_periods);
    free(h_attractors);

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

size_t read_input_file(std::string input_file,bool **initial_states){
    std::string line;
    std::ifstream myfile(input_file);
    size_t num_states = 0;

    if (myfile.is_open()) {
        while (getline(myfile, line)) {
            
            strtok((char *)line.c_str(), ",");
            strtok(NULL, ",");
            strtok(NULL, ",");
            char *str_num_states = strtok(NULL, ",");
            auto sz = std::stoul(str_num_states, nullptr, 10);
            num_states+=sz;            
        }
        myfile.clear();
        myfile.seekg(0);

        *initial_states = (bool *)malloc(sizeof(bool)*num_states*NUM_NOS);
        int num_nos_bytes = (int)ceil(NUM_NOS/8.0);
        int j = 0;
        while (getline(myfile, line)) {
            strtok((char *)line.c_str(), ",");
            char *init_state = strtok(NULL, ",");
            std::string init_state_str(init_state);
            int cb = 0;
            for(int i = 0,p = (num_nos_bytes*2)-2; i < num_nos_bytes;i++,p-=2){
                unsigned long v = std::stoul(init_state_str.substr(p,2), nullptr, 16);
                for(int b = 0;b < 8;b++){
                    (*initial_states)[j*NUM_NOS + cb] = (v & (1 << b));
                    cb++;
                    if(cb >= NUM_NOS){
                        i = num_nos_bytes;
                        break;
                    }
                }
            }
            j++;
        }
        myfile.close();
        return num_states;
    }
    else{
        std::cout << "Error: input file not found." << std::endl;
        exit(255);
    }

}
void write_output_file(std::string output_file, uint32_t * transients, uint32_t * periods, bool * attractors, size_t num_states){

    std::ofstream myfile(output_file);
    unsigned char b = 0;
    int cb = 0;
    for(unsigned long i = 0; i < num_states;i++){
            myfile <<  0 << "," << 0 <<  "," << periods[i] << "," <<  transients[i] <<  ",";
            for(int j=NUM_NOS-1; j >= 0 ;--j){
               b |= attractors[i*NUM_NOS + j];
               b <<= 1;
               cb++;
               if(cb == 8 || j == 0){
                     myfile << std::hex << std::setw(2) << std::setfill('0') << (int)b << std::dec;
                     b = 0;
                     cb = 0;
               }
            }
            myfile << std::endl;
    }
    myfile.close();
}

void write_report_file(double exec_time){
 std::ofstream myfile("performance_report.csv");
 myfile << "Execution time(ms)" << std::endl;
 myfile << exec_time << std::endl;
 myfile.close();
}
