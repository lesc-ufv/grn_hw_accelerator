#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define NUM_STATES (1<<20)
#define NUM_COPYS  (1 << 10)
#define NUM_NOS 70

typedef unsigned int uint32_t;

using namespace std; 

__device__
bool state_comp(bool *s0, bool *s1){
   for(int i = 0; i < NUM_NOS; i++){
       if(s0[i] != s1[i]) return false;
   }
   return true;
}

__device__
void pass(bool *aux){
  
  bool vet[NUM_NOS];
  for(int i = 0; i < NUM_NOS; i++){
       vet[i] = aux[i];
  }
  aux[0] = vet[41];
  aux[1] = vet[21];
  aux[2] = vet[41] && ! vet[33] ;
  aux[3] = vet[36] && ! ( vet[37] || vet[7] ) ;
  aux[4] = ( ( vet[51] || vet[34] ) && vet[37] ) && ! vet[3] ;
  aux[5] = ! ( vet[21] && vet[1] ) ;
  aux[6] = ( vet[49] || vet[32] ) && ! ( vet[34] || vet[37] ) ;
  aux[7] = ( vet[8] || vet[9] ) && ! vet[22] ;
  aux[8] = vet[18] && ! ( vet[11] || vet[33] ) ;
  aux[9] = vet[14] && ! ( vet[22] || vet[33] ) ;
  aux[10] = vet[47] && ! vet[48] ;
  aux[11] = vet[32] ;
  aux[12] = vet[43] && vet[53] ;
  aux[13] = ( vet[5] || vet[49] || vet[27] ) && ! vet[21] ;
  aux[14] = vet[31] ;
  aux[15] = vet[35] ;
  aux[16] = vet[29] ;
  aux[17] = vet[65] ;
  aux[18] = vet[53] || vet[17] ;
  aux[19] = vet[16] ;
  aux[20] = vet[60] ;
  aux[21] = ! ( vet[15] || vet[3] ) ;
  aux[22] = ( vet[32] || vet[49] ) && ! vet[44] ;
  aux[23] = ! vet[24] ;
  aux[24] = ( vet[3] || ( vet[43] && vet[53] ) ) ;
  aux[25] = vet[20] && ! vet[50] ;
  aux[26] = vet[2] || vet[30] ;
  aux[27] = ( ( vet[5] || vet[16] ) && vet[26] ) && ! vet[21] ;
  aux[28] = ( vet[34] && vet[3] ) && ! ( vet[21] || vet[0] ) ;
  aux[29] = vet[39] || vet[41] ;
  aux[30] = vet[10] || vet[52] || vet[53] ;
  aux[31] = ( vet[4] || vet[51] || vet[10] ) && ! vet[6] ;
  aux[32] = ! vet[23] ;
  aux[33] = ( vet[34] || vet[45] ) && ! ( vet[21] || vet[7] ) ;
  aux[34] = ( vet[38] || vet[26] || vet[0] ) && ! vet[28] ;
  aux[35] = vet[12] ;
  aux[36] = ( vet[15] || vet[40] ) && ! vet[38] ;
  aux[37] = vet[10] && ! vet[3] ;
  aux[38] = vet[34] && ! ( vet[32] || vet[27] ) ;
  aux[39] = vet[10] || vet[40] ;
  aux[40] = vet[15] || vet[20] ;
  aux[41] = vet[53] && ! vet[42] ;
  aux[42] = vet[32] || vet[49] ;
  aux[43] = vet[48] ;
  aux[44] = vet[31] ;
  aux[45] = vet[52] && ! vet[27] ;
  aux[46] = vet[45] || vet[32] ;
  aux[47] = vet[34] || vet[18] ;
  aux[48] = vet[16] || vet[53] ;
  aux[49] = vet[25] ;
  aux[50] = vet[49] ;
  aux[51] = vet[8] && ! vet[6] ;
  aux[52] = vet[58] && ! vet[46] ;
  aux[53] = vet[55] ;
  aux[54] = ( vet[63] || vet[66] ) && ! vet[60] ;
  aux[55] = vet[59] ;
  aux[56] = vet[61] && ! ( vet[64] || vet[58] ) ;
  aux[57] = ( vet[62] || vet[64] ) && ! ( vet[63] || vet[58] || vet[61] ) ;
  aux[58] = vet[54] ;
  aux[59] = ( vet[64] || vet[67] ) && ! vet[63] ;
  aux[60] = vet[59] || vet[66] || vet[32] ;
  aux[61] = vet[66] || vet[56] ;
  aux[62] = vet[66] || vet[59] ;
  aux[63] = vet[54] || vet[56] ;
  aux[64] = vet[57] || vet[65] ;
  aux[65] = vet[64] && ! vet[58] ;
  aux[66] = ( vet[67] || vet[55] ) && ! vet[63] ;
  aux[67] = vet[32] ;
  aux[68] = ( vet[19] && vet[13] ) && ! ( vet[33] || vet[7] ) ;
  aux[69] = vet[7] ;

}

__global__
void findAttractor(bool *attractors, uint32_t * transients, uint32_t *periods, uint32_t numThreads){
   const int numNos = NUM_NOS;
   const int numState = NUM_STATES;
   const int numCopys = NUM_COPYS;
   unsigned int periodo, transient;
   bool S0[numNos];
   bool S1[numNos];
   uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   uint32_t step =  numState / numCopys; 
   uint32_t rest =  numState % numCopys;
   uint32_t begin = 0;
   uint32_t end = step - 1;
   bool flag = true;
   if(thread < numThreads){
       if(rest > 0){
           end = end + 1;
           rest = rest - 1;
       }else{
           flag = false;
       }
       
       for(uint32_t i = 0; i < numCopys;i++){
         if(i == thread) break;
         if(rest > 0){
           end = end + 1;
           begin = begin + 1;
           rest = rest - 1;
         }
         else if(rest == 0 && flag){
           begin = begin + 1;
           flag = 0;
         }
         begin += step;
         end += step;
       }
       //printf("Thread %d: begin:%d end: %d numStates: %d\n",thread,begin,end,end-begin+1);
       for(uint32_t  i = begin; i <= end; i++){
          int aux = i;
          for(int k = 0; k < numNos; k++){
              S0[k] = aux & 1 != 0;
              S1[k] = aux & 1 != 0;
	      aux >>= 1;
          }
          periodo = 0;
          transient = 0;
          do{
             pass(S0);
             pass(S1);
             pass(S1);
             transient++;
          }while (state_comp(S0,S1));
          do{
             pass(S0);
             periodo++;
          }while(state_comp(S0,S1));
          periodo--;
	  
          transients[i] = transient;
          periods[i]= periodo;
          for(int s = 0; s < numNos; s++){
            attractors[i*numNos + s] = S0[s];
	  }
       }
   }
}


int getIndice(int num_col, int i, int j)
{
    return (i*num_col)+ j;
}
/**
 * Host main routine
 */
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    size_t numNos = NUM_NOS;
    size_t numState = NUM_STATES;
    size_t size = numNos * numState * sizeof(bool);
    size_t size_transients = numState*sizeof(uint32_t);
    size_t size_periods = numState*sizeof(uint32_t);
    size_t totalBytes = size+size_transients+size_periods;
    size_t kb = totalBytes/(1024);
    size_t mb = kb/(1024);
    size_t gb = mb/(1024);
    printf("Find attractors net %lu nodes in %lu initials states.\n", numNos,numState);
    printf("Memory usage: %lu Gb or %lu Mb or %lu Kb.\n",gb,mb,kb);
    uint32_t *h_transients = (uint32_t*)malloc(size_transients);
    // Verify that allocations succeeded
    if (h_transients == NULL)
    {
        fprintf(stderr, "Failed to allocate h_transients!\n");
        exit(EXIT_FAILURE);
    }    
    uint32_t *h_periods = (uint32_t*)malloc(size_periods);
    // Verify that allocations succeeded
    if (h_periods == NULL)
    {
        fprintf(stderr, "Failed to allocate h_periods!\n");
        exit(EXIT_FAILURE);
    }
    // Allocate the host output vector C
    bool *h_attractors = (bool *)malloc(size);

    // Verify that allocations succeeded
    if (h_attractors == NULL)
    {
        fprintf(stderr, "Failed to allocate h_attractors!\n");
        exit(EXIT_FAILURE);
    }
    // Allocate the device vectors
    uint32_t *d_transients = NULL;
    err = hipMalloc((void **)&d_transients, NUM_STATES*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    uint32_t *d_periods = NULL;
    err = hipMalloc((void **)&d_periods, NUM_STATES*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    bool *d_attractors = NULL;
    err = hipMalloc((void **)&d_attractors, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid =(NUM_COPYS + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    findAttractor<<<blocksPerGrid, threadsPerBlock>>>(d_attractors,d_transients,d_periods,NUM_COPYS);
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch findAttractor kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_transients, d_transients,size_transients, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_transients from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
        err = hipMemcpy(h_periods, d_periods, size_periods, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_periods from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_attractors, d_attractors, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_attractors from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*
    printf("Attractor found:\n");
    for(int i = 0; i < numState; i++){
       for(int j = 0; j < numNos; j++){
          printf("%d",h_attractors[getIndice(numNos,i,j)]);
      }
      printf("\n");
    }
    printf("\n");
    */
    err = hipFree(d_transients);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_transients (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_periods);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_periods (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_attractors);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_attractors (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_transients);
    free(h_periods);
    free(h_attractors);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}





